#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
// #include <opencv2/core/core.hpp>
// #include <opencv2/highgui/highgui.hpp>
// #include <opencv2/opencv.hpp>
#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudaarithm.hpp"
#include <vector>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

size_t numRows, numCols;

/* Mirror operations */

__global__ 
void mirror(const uchar4* const inputChannel, uchar4* outputChannel, int numRows, int numCols, bool vertical)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if ( col >= numCols || row >= numRows )
  {
   return;
  }

  if(!vertical)
  { 
  
    int thread_x = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_y = blockDim.y * blockIdx.y + threadIdx.y;
    
    int thread_x_new = thread_x;
    int thread_y_new = numRows-thread_y;

    int myId = thread_y * numCols + thread_x;
    int myId_new = thread_y_new * numCols + thread_x_new;
    outputChannel[myId_new] = inputChannel[myId];
   	
  }

  else
  {
  	int thread_x = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_y = blockDim.y * blockIdx.y + threadIdx.y;
    
    int thread_x_new = numCols-thread_x;
    int thread_y_new = thread_y;

    int myId = thread_y * numCols + thread_x;
    int myId_new = thread_y_new * numCols + thread_x_new;
  
  	outputChannel[myId_new] = inputChannel[myId];  // linear data store in global memory	
  }
}         



uchar4* mirror_ops(uchar4 *d_inputImageRGBA, size_t numRows, size_t numCols, bool vertical)
{
	//Set reasonable block size (i.e., number of threads per block)
    const dim3 blockSize(4,4,1);
    //Calculate Grid SIze
    int a=numCols/blockSize.x, b=numRows/blockSize.y;	
    const dim3 gridSize(a+1,b+1,1);

    const size_t numPixels = numRows * numCols;

    uchar4 *d_outputImageRGBA;
    hipMalloc(&d_outputImageRGBA, sizeof(uchar4) * numPixels);

    //Call mirror kernel.
    mirror<<<gridSize, blockSize>>>(d_inputImageRGBA, d_outputImageRGBA, numRows, numCols, vertical);

    hipDeviceSynchronize(); 
    
    //Initialize memory on host for output uchar4*
    uchar4* h_out;
    h_out = (uchar4*)malloc(sizeof(uchar4) * numPixels);

    //copy output from device to host
    hipMemcpy(h_out, d_outputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost);
    
    //cleanup memory on device
    hipFree(d_inputImageRGBA);
    hipFree(d_outputImageRGBA);

    //return h_out
        return h_out;
}


void loadImageRGBA(string &filename, uchar4 **imagePtr,size_t *numRows, size_t *numCols)
{
    // loading the image
    cv::Mat image = imread(filename.c_str(), cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);

    // forming a 4-channel(RGBA) image.
    cv::Mat imageRGBA;
    cvtColor(image, imageRGBA, cv::COLOR_BGR2BGRA);

    *imagePtr = new uchar4[image.rows * image.cols];
    unsigned char *cvPtr = imageRGBA.ptr<unsigned char>(0);
    for(size_t i = 0; i < image.rows * image.cols; ++i)
    {
        (*imagePtr)[i].x = cvPtr[4*i + 0];
        (*imagePtr)[i].y = cvPtr[4*i + 1];
        (*imagePtr)[i].z = cvPtr[4*i + 2];
        (*imagePtr)[i].w = cvPtr[4*i + 3];
    }
    *numRows = image.rows;
    *numCols = image.cols;
}

void saveImageRGBA(uchar4* image, string &output_filename,size_t numRows, size_t numCols)
{
    // Forming the Mat object from uchar4 array.
    int sizes[2] = {numRows, numCols};
    Mat imageRGBA(2, sizes, CV_8UC4, (void *)image);
    // Converting back to BGR system
    Mat imageOutputBGR;
    cvtColor(imageRGBA, imageOutputBGR, cv::COLOR_BGR2BGRA);
    // Writing the image
    imwrite(output_filename.c_str(), imageOutputBGR);
}

uchar4* load_image_in_GPU(string filename)
{ // Load the image into main memory
  uchar4 *h_image, *d_in;
  loadImageRGBA(filename, &h_image, &numRows, &numCols);
  // Allocate memory to the GPU
  hipMalloc((void **) &d_in, numRows * numCols * sizeof(uchar4));
  hipMemcpy(d_in, h_image, numRows * numCols * sizeof(uchar4), hipMemcpyHostToDevice);
  // No need to keep this image in RAM now.
  free(h_image);
  return d_in;
}




int main(int argc, char **argv)
{
    // variables_map vm;
    string input_file = "image.png";
    string output_file = "output.png";
    
    uchar4 *d_in = load_image_in_GPU(input_file);
    uchar4 *h_out = NULL;

    h_out = mirror_ops(d_in, numRows, numCols, true);

    hipFree(d_in);
    if(h_out != NULL)
    {
        saveImageRGBA(h_out, output_file, numRows, numCols); 
    }
        
}