
#include <hip/hip_runtime.h>
#include <stdio.h>
#define T 64 // As Threads
#define array_size 256

__global__ void vecMultiplyReverse(int *A, int *B) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int Reverse = (T - 1) - i;
	B[Reverse] = A[i];
    
}
int main (int argc, char *argv[])
{
	int i;
	int size = T*sizeof(int);
    int a[T],b[T], *devA,*devB;
	for (i=0; i< T; i++)
	{
		a[i] = i + 1; 	
	}
	
	hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    hipMemcpy( devB, b, size, hipMemcpyHostToDevice);
    dim3 dimBlock(T);
    dim3 dimGrid(array_size/T - 1);
    vecMultiplyReverse<<<dimGrid,dimBlock>>>(devA,devB);
	printf("Before\n");
	for (i=0; i< T; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

    hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
	printf("After\n");
    for (i=0; i < T; i++) 
    {
		printf("%d ",b[i]);
	}
	printf("\n");
return 0;
}


