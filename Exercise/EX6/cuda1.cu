
#include <hip/hip_runtime.h>
#include <stdio.h>
#define T 16 // As Threads
#define array_size 64

__global__ void vecMultiply(int *A) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	A[i] = A[i] * 2;
}

int main (int argc, char *argv[])
{
	int i;
	int size = T*sizeof(int);
    int a[array_size], *devA;
	for (i=0; i< array_size; i++)
	{
		a[i] = i + 1; 	
	}
	hipMalloc( (void**)&devA,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    dim3 dimBlock(T,T);
    dim3 dimGrid(array_size/T - 1);
    vecMultiply<<<dimGrid,dimBlock>>>(devA);
	printf("Before\n");
	for (i=0; i< array_size; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	printf("After\n");
    for (i=0; i < array_size; i++)
    {
		printf("%d ",a[i]);
	}
	printf("\n");

    return 0;
}


