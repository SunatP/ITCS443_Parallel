
#include <hip/hip_runtime.h>
#include <stdio.h>
#define T 16 // As Threads
#define array_size 64

__global__ void vecMultiplyReverse(int *A, int *B, int *C) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i%2 == 0)
    {
        C[i] = A[i] + B[i];
    }
    else if(i%2 != 0)
    {
        C[i] = A[i] - B[i];
    }
}

int main (int argc, char *argv[])
{
	int i;
	int size = T*sizeof(int);
    int a[T],b[T],c[T], *devA,*devB,*devC;
	for (i=0; i< T; i++)
	{
        a[i] = i + 2;
        b[i] = i + 1; 
	}
	
	hipMalloc( (void**)&devA,size);
    hipMalloc( (void**)&devB,size);
    hipMalloc( (void**)&devC,size);
    hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
    hipMemcpy( devB, b, size, hipMemcpyHostToDevice);
    hipMemcpy( devC, c, size, hipMemcpyHostToDevice);
    dim3 dimBlock(T);
    dim3 dimGrid(array_size/T - 1);
    vecMultiplyReverse<<<dimGrid,dimBlock>>>(devA,devB,devC);
	printf("Before A: \n");
	for (i=0; i< T; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

    printf("Before B: \n");
	for (i=0; i< T; i++)
	{
		printf("%d ", b[i]);	
	}	
	printf("\n");

    hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
    hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
	printf("After\n");
    for (i=0; i < T; i++) 
    {
		printf("%d ",c[i]);
	}
	printf("\n");
return 0;
}


