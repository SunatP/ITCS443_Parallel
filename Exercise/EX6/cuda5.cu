
#include <hip/hip_runtime.h>
#include <stdio.h>
#define T 8 // As Threads
#define N 16


__global__ void vecMatrix(int *A, int *B) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y+ threadIdx.y;
    int width = gridDim.x * T;
    for( int j = 0; j<T; j+=N )
    {
        B[x*width + (j+y)] = A[(y+j)*width + x];
    }

}

int main (int argc, char *argv[]){
	int i,j;
    int size[N*N];
    int A[N][N];

    int sizearr = N*N *sizeof(int);

    int *Adefault,*B;

	for (i=0; i< N; i++)
	{
        for(j = 0 ; j<N ; j++ )
        {
            A[i][j] = ((i*i) +1) * (j+1);
            printf("%5d ", A[i][j]);
        }
    }
    printf("\n");

	hipMalloc( (void**)&Adefault,sizearr);
    hipMalloc( (void**)&B,sizearr);
    hipMemcpy( Adefault, A, sizearr, hipMemcpyHostToDevice);

    dim3 dimBlock(T,T);
    dim3 dimGrid((N+ dimBlock.x - 1)/ dimBlock.x ,(N + dimBlock.y - 1) / dimBlock.y);
    vecMatrix<<<dimGrid,dimBlock>>>(Adefault,B);
    hipMemcpy(size, B, sizearr, hipMemcpyDeviceToHost);
    hipFree(Adefault);
    hipFree(B);

    printf("Result\n");
    int newline = 0;

	for (i=0; i < N * N; i++) {
        newline++;
        printf("%3d ",size[i]);
        if(newline == N)
        {
            newline = 0;
            printf("\n");
        }	
	}
	printf("\n");

}


