
#include <hip/hip_runtime.h>
#include <stdio.h>
#define T 8 // As Threads
#define N 16


__global__ void vecMatrix(int *A, int *B) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int id = (i * N) + j;
    if(i < N && j < N)
    {
        B[id] = A[id] + 1;
    }
}

int main (int argc, char *argv[])
{
	int i,j;
    int size[N*N];
    int A[N][N];

    int sizearr = N*N *sizeof(int);

    int *Adefault,*B;

	for (i=0; i< N; i++)
	{
        for(j = 0 ; j<N ; j++ )
        {
            A[i][j] = ((i*i) +1) * (j+1);
            printf("%5d ", A[i][j]);
        }
    }
    printf("\n");
	hipMalloc( (void**)&Adefault,sizearr);
    hipMalloc( (void**)&B,sizearr);
    hipMemcpy( Adefault, A, sizearr, hipMemcpyHostToDevice);

    dim3 dimBlock(T,T);
    dim3 dimGrid((N+ dimBlock.x - 1)/ dimBlock.x ,(N + dimBlock.y - 1) / dimBlock.y);
    vecMatrix<<<dimGrid,dimBlock>>>(Adefault,B);
    hipMemcpy(size, B, sizearr, hipMemcpyDeviceToHost);
    hipFree(Adefault);
    hipFree(B);
	printf("Result\n");
    for (i=0; i < N * N; i++)
    {
		printf("%5d ",size[i]);
	}
	printf("\n");
return 0;
}


