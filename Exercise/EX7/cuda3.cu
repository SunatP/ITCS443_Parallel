
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define n 16

__global__ void countNumberInArray(int *originalData, int *arrayCount)
{    
    int index = blockIdx.x * blockDim.x + threadIdx.x, i;
    int sum = 0;
    if(threadIdx.x < n)
    {
    for(i = 0; i < n; i++)
    {
        if(i < n)
        {
            sum += originalData[(index * n) + i];
            // atomicAdd(&arrayCount[index],sum);
        }
        else
        {
            sum += originalData[(index * n) + index];
            // atomicAdd(&arrayCount[index],sum);
        }
        printf("%3d " ,threadIdx.x);
    }
    printf("\n");
    }
    
    else
    {
        for(i = 0; i < n; i++)
        {
            if(i < n)
            {
                sum += originalData[(index * n) + i];
                // atomicAdd(&arrayCount[index],sum);
            }
            else
            {
                sum += originalData[(index * n) + index];
                // atomicAdd(&arrayCount[index],sum);
            }
            printf("%3d " ,threadIdx.x);
        }  

    }
 
    atomicAdd(&arrayCount[index],sum);
}

int main(int argc, char *argv[])
{
    int totalCount = 2 * n;
    int originalData[n][n], count[totalCount];
    int i, j;
    int *deviceOriginalData, *deviceArrayCount;
    int arrayByteSize = (n *n) * sizeof(int);
    int countArrayByteSize = totalCount * sizeof(int);
    printf("\n"); 
    printf("ORIGINAL: \n");
    for(i = 0; i < n; i++)
    {
        for(j = 0; j < n; j++)
        {
            originalData[i][j] = i;
            printf("%3d ", originalData[i][j]);
        }
        printf("\n");
    }
    printf("\n\n");

    hipMalloc((void**) &deviceOriginalData, arrayByteSize);
    hipMalloc((void**) &deviceArrayCount, countArrayByteSize);
    hipMemcpy(deviceOriginalData, originalData, arrayByteSize, hipMemcpyHostToDevice);
    
    dim3 blockDim(totalCount);
    countNumberInArray<<<1, blockDim>>>(deviceOriginalData, deviceArrayCount);
    
    hipMemcpy(count, deviceArrayCount, countArrayByteSize, hipMemcpyDeviceToHost);
    hipFree(deviceOriginalData);
    hipFree(deviceArrayCount);

    int rowCounts[n], colCounts[n], rowArrayIterator = 0, colArrayIterator = 0;
    int rowsum = 0;
    int colsum = 0;
    int l = 0;
    for(l = 0; l < totalCount; l++)
    {
        if(l < n)
        {
            rowCounts[rowArrayIterator++] = count[l]; 
            rowsum += count[l];
        } 
    }
    for(l = 0; l < totalCount; l++)
    {
        if(l < n)
        {
            colCounts[colArrayIterator++] = count[l];     
        }
        colsum += count[l];
    }
    printf("\nTOTAL COUNT ROW\n");
    for(l = 0; l < n; l++)
    {
        printf("(%d,%3d)", l, rowCounts[l]);
    }
    printf("\nSum Row: %d\n" ,rowsum);
    printf("\n\nTOTAL COUNT COL\n");
    for(l = 0; l < n; l++)
    {
        printf("(%d,%3d)", l, colCounts[l]);
    }
    printf("\nSum Col: %d\n" ,colsum);
    printf("\n");
    return 0;
}