
#include <hip/hip_runtime.h>
#include <stdio.h>

#define n 16

__global__ void countNumberInArray(int *originalData, int *arrayCount)
{    
    int index = threadIdx.x, i;
    int sum = 0;
    if(threadIdx.x < n)
    {
        for(i = 0; i < n; i++)
        {
            sum += originalData[(index * n) + i];
            printf("%3d " ,threadIdx.x);
        }
    }
    else
    {
        for(i = 0; i < n; i++)
        {
            sum += originalData[(i * n) + index];
            printf("%3d " ,threadIdx.x);
        }   
    }
    atomicAdd(&arrayCount[index],sum);        
}

int main(int argc, char *argv[])
{

    int totalCount = 2 * n;
    int originalData[n][n], count[totalCount];
    int i = 0;
    int j = 0;

    int *deviceOriginalData, *deviceArrayCount;

    int arrayByteSize = (n * n) * sizeof(int);
    int countArrayByteSize = totalCount * sizeof(int);

    printf("ORIGINAL: \n");
    for(i = 0; i < n; i++)
    {
        for(j = 0; j < n; j++)
        {
            originalData[i][j] = i;
            printf("%3d ", originalData[i][j]);
        }
        printf("\n");
    }
    printf("\n\n");

    hipMalloc((void**) &deviceOriginalData, arrayByteSize);
    hipMalloc((void**) &deviceArrayCount, countArrayByteSize);
    hipMemcpy(deviceOriginalData, originalData, arrayByteSize, hipMemcpyHostToDevice);
    
    dim3 blockDim(totalCount);
    countNumberInArray<<<1, blockDim>>>(deviceOriginalData, deviceArrayCount);
    
    hipMemcpy(count, deviceArrayCount, countArrayByteSize, hipMemcpyDeviceToHost);
    hipFree(deviceOriginalData);
    hipFree(deviceArrayCount);

    int rowCounts[n], colCounts[n], rowArrayIterator = 0, colArrayIterator = 0;
    int rowsum = 0;
    int colsum = 0;
    int l = 0;
    for(l = 0; l < totalCount; l++)
    {
        if(l < n)
        {
            rowCounts[rowArrayIterator++] = count[l];
            rowsum += count[l];
        }
        else
        {
            colCounts[colArrayIterator++] = count[l];
            colsum += count[l];
        }
    }
    printf("TOTAL COUNT ROW\n");
    for(l = 0; l < n; l++)
    {
        printf("(%d,%3d)", l, rowCounts[l]);
    }
    printf("\nSum Row: %d\n" ,rowsum);
    printf("\n\nTOTAL COUNT COL\n");
    for(l = 0; l < n; l++)
    {
        printf("(%d,%3d)", l, colCounts[l]);
    }
    printf("\nSum Col: %d\n" ,colsum);
    printf("\n");
    return 0;
}