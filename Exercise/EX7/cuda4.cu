
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define T 256
#define n 1024

__global__ void reduceToSummation(int *originalData, int stride)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idx = 2 * stride * threadId;
    if(idx < n)
    {
        originalData[idx] = originalData[idx] + originalData[idx + stride];
    }
}

__global__ void reduceToMinimum(int *originalData, int stride)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idx = 2 * stride * threadId;
    if(idx < n)
    {
        int min = originalData[idx];
        if(originalData[idx + stride] < min)
        {
            min = originalData[idx + stride];
        }
        originalData[idx] = min;
    }
}

__global__ void reduceToMaximum(int *originalData, int stride)
{
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idx = 2 * stride * threadId;
    if(idx < n)
    {
        int max = originalData[idx];
        if(originalData[idx + stride] > max)
        {
            max = originalData[idx + stride];
        }
        originalData[idx] = max;
    }
}

int main(int argc, char *argv[])
{
    int originalData[n];
    int sum, min, max;
    int i;
    int *deviceOriginalData;
    int arrayByteSize = n * sizeof(int);
    printf("ORIGINAL: \n");
    for(i = 0; i < n; i++)
    {
        originalData[i] = i;
        printf("%3d ", originalData[i]);
    }
    printf("\n\n"); 
    // Allocates Once for all kernels
    hipMalloc((void**) &deviceOriginalData, arrayByteSize);
   
    // KERNEL 1: Find Average by Finding Summation
    hipMemcpy(deviceOriginalData, originalData, arrayByteSize, hipMemcpyHostToDevice);
    for(int s = 1; s < n; s *= 2)
    {
        reduceToSummation<<<(n + T - 1) / T, T>>>(deviceOriginalData, s);
    }
    hipMemcpy(&sum, deviceOriginalData, sizeof(int), hipMemcpyDeviceToHost);
    double realAverage = sum / (double) n;
    
    // KERNEL 2: Find Minimum
    hipMemcpy(deviceOriginalData, originalData, arrayByteSize, hipMemcpyHostToDevice);
    for(int s = 1; s < n; s *= 2)
    {
        reduceToMinimum<<<(n + T - 1) / T, T>>>(deviceOriginalData, s);
    }
    hipMemcpy(&min, deviceOriginalData, sizeof(int), hipMemcpyDeviceToHost);

    // KERNEL 3: Find Maximum
    hipMemcpy(deviceOriginalData, originalData, arrayByteSize, hipMemcpyHostToDevice);
    for(int s = 1; s < n; s *= 2)
    {
        reduceToMaximum<<<(n + T - 1) / T, T>>>(deviceOriginalData, s);
    }
    hipMemcpy(&max, deviceOriginalData, sizeof(int), hipMemcpyDeviceToHost);

    // Free the memory
    hipFree(deviceOriginalData);

    // Print the results
    printf("\nAverage is %.2f", realAverage);
    printf("\nThe Minimum Number is %d\n", min);
    printf("The Maximum Number is %d\n", max);
    return 0;
}