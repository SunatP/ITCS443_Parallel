#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/sort.h>
#define thread 512
__global__ void rank_sort(int *data, int *result)
{
    int i,j,position;
    position = 0;
    i = threadIdx.x;
    int self = data[i];

    for(j = 0; j < thread; j++)
    {
        if(( self > data[j]) || (self == data[j]) && (j < i))
        {
          position+=1;
        }      
    }
    result[position] = self;
}   

int main(int argc, char *argv[]){
  int *arr,*data;
  int i;
  int Data[thread],sort[thread];
  int size =  sizeof(int)*thread;
//   (float *)malloc(blocks*threads_per_block*sizeof(float));
  srand(123);
  printf(" Generate Ok\n");
  for(i = 0; i < thread; i++)
  {
     Data[i] = rand() % 100;
     printf("%d ",Data[i]);
  }
  printf(" \n Working Ok\n");
  hipMalloc( (void**) &arr, size);
  hipMalloc( (void**) &data, size);
  printf(" Mallock Ok\n");
  // thrust::sort(arr, arr + size);
  hipMemcpy(arr,Data,size, hipMemcpyHostToDevice);
  printf(" Copy Ok\n");
  rank_sort<<<1,thread>>>(arr,data);
  printf(" Function Ok\n");
  hipMemcpy(sort,data,size,hipMemcpyDeviceToHost);
  printf(" Copy Back Ok\n");

  printf(" Sorted Data \n");
  for(i = 0 ; i < thread ; i++)
  {
    printf("%d ",sort[i]);
  }
  printf("\n");
  printf(" Sorted OK \n");
  hipFree(data);  
  hipFree(arr);
  return 0;
}
