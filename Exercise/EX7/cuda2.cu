#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/sort.h>
#define num_thread 64
#define thread 16
__global__ void count(int *data,int input, int *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(data[i] == input)
    {
        int a = 1;
        atomicAdd(result,a);
    }
}   

int main(int argc, char *argv[]){
  int Data[num_thread], *arr,input,*result;
  int i;
  int resultarr[1];
  int size = sizeof(int)*num_thread;
  srand(123456846);
  printf(" Generate Ok\n");
  hipSetDevice(0);
  for(i = 0; i < num_thread; i++)
  {
     Data[i] = rand() % 50;
     printf("%d ",Data[i]);
  }
  printf(" \n Working Ok\n");
  printf("Input value to find: ");
  scanf("%d",&input);
  hipMalloc( (void**) &arr, size);
  hipMalloc( (void**) &result, sizeof(int));
  printf(" Malloc Ok\n");

  hipMemcpy(arr,Data,size, hipMemcpyHostToDevice);
  
  printf(" Copy Ok\n");
  
  count<<<num_thread/thread,thread>>>(arr,input,result);
  
  printf(" Function Ok\n");
  
  hipMemcpy(resultarr,result,sizeof(int),hipMemcpyDeviceToHost);
  printf(" Copy Back Ok\n");
  
  hipFree(result);  
  hipFree(arr);
  printf(" Value %d to search occurrences Data found: %d",input,resultarr[0]);

  printf("\n");

  return 0;
}
