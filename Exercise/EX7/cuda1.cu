#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/sort.h>
#define array_size 64
#define thread 16
__global__ void rank_sort(int *data, int *result)
{
    int i,j,position;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    int self = data[i];
    for(;i < array_size; i++)
    {
        position = 0;
    for(j = 0; j < array_size; j++)
        if(( self > data[j]) || (self == data[j]) && (j < i))
            position+=1;
    result[position] = self;
    }
}   

int main(int argc, char *argv[]){
  int *arr = (int *) malloc(sizeof(int)*array_size);
  int i;
  int *data = (int *) malloc(sizeof(int)*array_size);
  int *result = (int *) malloc(sizeof(int)*array_size);
  int size =  sizeof(int)*thread;
//   (float *)malloc(blocks*threads_per_block*sizeof(float));
  srand(123);
  for(i = 0; i < array_size; i++)
  {
     arr[i] = rand() % 50;
  }
  printf(" Working Ok\n");
  hipMalloc( (void**) &data, thread*array_size);
  hipMalloc( (void**) &result, thread*array_size);
  printf(" Mallock Ok\n");
  thrust::sort(arr, arr + size);
  hipMemcpy(data,arr,size, hipMemcpyHostToDevice);
  printf(" Copy Ok\n");
  dim3 dimBlock(thread);
  dim3 dimGrid(array_size/thread - 1);
  rank_sort<<<dimGrid,dimBlock>>>(data,result);
  printf(" Function Ok\n");
  hipMemcpy(arr,data,size,hipMemcpyDeviceToHost);
  printf(" Copy Back Ok\n");

  printf(" Sorted Data \n");
  for(i = 0 ; i < array_size ; i++)
  {
    printf("%d ",arr[i]);
  }
  printf("\n");
  printf(" Sorted OK \n");
  hipFree(data);  
  hipFree(result);
  return 0;
}
