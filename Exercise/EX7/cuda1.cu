#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define thread 256
#define arraySize 64
/**
 From the following sequential Rank sort algorithm that allows duplicate numbers, 
 implement a parallel version in CUDA. 
**/

__global__ void RankSort(int *data, int *result)
{
   int i,j,position;
   
   i = threadIdx.x;
   int self = data[i];

   for(j = 0 ; j < arraySize ; j++)
   {
       position = 0;
       if((self > data[i]) || (self == data[j]) && (j < i))
       {
           position++;
       }
       result[position] = self;
   }
}
int main(int argc,char *argv[])
{
    int arr[arraySize];
    int i;
    int size = thread*sizeof(int);
    int *data, *result;
    srand(time(0));
    for(i = 0 ; i < arraySize ; i++)
    {
         arr[i] = rand() % 50;
    }
    hipMalloc( (void**)&data,size);
    hipMalloc( (void**)&result,size);
	hipMemcpy( data, arr, size, hipMemcpyHostToDevice);

    RankSort<<<thread/arraySize,thread>>>(data,result);

    hipMemcpy( arr, result,size,hipMemcpyDeviceToHost);
    hipFree(data);
    hipFree(result);

    printf("\n sorted data");
    for(i = 0 ; i< arraySize ;i++)
    {
        printf("%d", data[i]);
    }
    printf("\n");
    return 0;
}