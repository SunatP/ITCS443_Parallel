
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA to assign a value to each element of the array of integers A[256] using 256 threads. 
// Each A[i] is assigned with the value of 2*i, for  i = 0 to 255.
#define T 256 // As Threads

__global__ void reverseArray(int *A, int *B) {
	int threadID = threadIdx.x;
	int Reverse = (T - 1) - threadID;
	B[Reverse] = A[threadID];
}

int main (int argc, char *argv[]){
	int i;
	int size = T*sizeof(int);
	int a[T],b[T], *devA,*devB;
	for (i=0; i< T; i++)
	{
		a[i] = i + 1; 	
	}
	
	hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

	
	reverseArray<<<1, T>>>(devA,devB); // 1 , 256 mean send each data with total thread 256 threads
	printf("Before\n");
	for (i=0; i< T; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	printf("After\n");
	for (i=0; i < T; i++) {
		printf("%d ",b[i]);
	}
	printf("\n");

}


