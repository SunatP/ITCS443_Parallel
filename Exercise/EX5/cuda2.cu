
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA to assign a value to each element of the array of integers A[256] using 256 threads. 
// Each A[i] is assigned with the value of 2*i, for  i = 0 to 255.
#define T 256 // As Threads
#define ArraySize 1314
// #define n 256

__global__ void vecMultiply(int *A) {
	int i;
	int threadID = threadIdx.x;
	int start = (threadID * ArraySize) / 256;
	int end = ( ( (threadID + 1 ) * ArraySize) / 256) - 1;
	for(i = start ; i < end ; i++)
	{
		A[i] = A[i] * 2;
	}
}

int main (int argc, char *argv[]){
	int i;
	int size = ArraySize*sizeof(int);
	int a[size], *devA;
	for (i=0; i< ArraySize; i++)
	{
		a[i] = i + 1; 	
	}
	
	hipMalloc( (void**)&devA,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);


	
	vecMultiply<<<1, 256>>>(devA); // 1 , 256 mean send each data with total thread 256 threads
	printf("Before\n");
	for (i=0; i< ArraySize; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	printf("After\n");
	for (i=0; i < ArraySize; i++) {
		printf("%d ",a[i]);
	}
	printf("\n");

}


