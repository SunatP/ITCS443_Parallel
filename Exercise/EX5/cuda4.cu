
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA to assign a value to each element of the array of integers A[256] using 256 threads. 
// Each A[i] is assigned with the value of 2*i, for  i = 0 to 255.
#define T 256 // As Threads
#define ArraySize 1314

__global__ void reverseArray(int *A, int *B) {
	int threadID = threadIdx.x;
	int start = (threadID * ArraySize) / 256;
	int end = ( ( (threadID + 1 ) * ArraySize) / 256) - 1;
	while(end > 0)
	{
		B[end] = A[start];
		end--;
		start++;
	}
}

int main (int argc, char *argv[]){
	int i;
	int size = ArraySize*sizeof(int);
	int a[ArraySize],b[ArraySize], *devA,*devB;
	for (i=0; i< ArraySize; i++)
	{
		a[i] = i + 1; 	
	}
	
	hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

	
	reverseArray<<<1, 256>>>(devA,devB); // 1 , 256 mean send each data with total thread 256 threads
	printf("Before\n");
	for (i=0; i< ArraySize; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipMemcpy(b, devB, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	printf("After\n");
	for (i=0; i < ArraySize; i++) {
		printf("%d ",b[i]);
	}
	printf("\n");

}


