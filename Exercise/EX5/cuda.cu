
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA to assign a value to each element of the array of integers A[256] using 256 threads. 
// Each A[i] is assigned with the value of 2*i, for  i = 0 to 255.
#define T 256 // As Threads
// #define n 256

__global__ void vecMultiply(int *A) 
{
	int i = threadIdx.x;
	A[i] = A[i] * 2;
}

int main (int argc, char *argv[])
{
	int i;
	int size = T*sizeof(int);
	int a[T], *devA;
	for (i=0; i< T; i++)
	{
		a[i] = i + 1; 	
	}
	
	hipMalloc( (void**)&devA,size);
	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);

	vecMultiply<<<1, T>>>(devA);
	printf("Before\n");
	for (i=0; i< T; i++)
	{
		printf("%d ", a[i]);	
	}	
	printf("\n");

	hipMemcpy(a, devA, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	printf("After\n");
	for (i=0; i < T; i++) {
		printf("%d ",a[i]);
	}
	printf("\n");

}


