
#include <hip/hip_runtime.h>
#include <stdio.h>
#define T 256
#define n 1024
// #define n 1029 

__global__ void vecAdd(float *A, float *B, float *C) {
    int i;
    
    i = blockIdx.x*blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
    
}

int main (int argc, char *argv[]){
    int i; 
    int size = n * sizeof(float);
    float a[n] , b[n], c[n], *devA , *devB,* devC;
    for(i=0 ; i <n ; i++)
    {
        a[i] = 1 ; b[i] = 2;

    }
    hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
	hipMalloc( (void**)&devC,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

    int nblocks = n/T;
    // int nblocks = (n+T - 1) / T; // Efficient way to run

	vecAdd<<<nblocks, T>>>(devA, devB, devC);

	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

    for(i = 0 ; i< n ; i++)
    {
        printf("%f",c[i]);

    }
    printf("\n");
}



