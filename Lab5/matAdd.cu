#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 16

__global__ void addMatrix(float *A, float *B, float *C) {
    int i = threadIdx.y;
    int j = threadIdx.x;
    int index = i*N+j;
    C[indexi] = A[index] + B[index];
}

int main(int argc , char *argv[])
{
    int i,j;
    int size = N * N * sizeof(float);
    float = a[N][N],b[N][N], c[N][N], *devA, *devB,*devC;

    for(i=0; i<N;i++)
    {
        for(j=0;j< N ;j++)
        {
            a[i][j] = 1;b[i][j]=2;
        }
    }
    hipMalloc( (void**)&devA,size);
	hipMalloc( (void**)&devB,size);
	hipMalloc( (void**)&devC,size);

	hipMemcpy( devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy( devB, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock (N,N);
    dim3 dimGrid (1,1);

    // int nblocks = n/T;
    // int nblocks = (n+T - 1) / T; // Efficient way to run

	addMatrix<<<dimGrid, dimBlock>>>(devA, devB, devC);

	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
    hipFree(devC);
    for(i=0; i<N;i++)
    {
        for(j=0;j< N ;j++)
        {
           printf("%.2f " ,c[i][j]);
        }
        printf("\n");
    }   
}